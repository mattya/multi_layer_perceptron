
#include <iostream>
#include <vector>
#include <fstream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <hipblas.h>

#include "kernel_misc.h"
#include "prop.h"
#include "file_manager.h"
#include "memory_alloc.h"
#include "train.h"

using namespace std;

float beta = 10.0;
float lambda = 0.00000001;
float eta0 = 0.015;
int pitch_x = 32, pitch_y = 32;

int batch_size = 1000;
int NTrain = 10000;
int NTest = 10000;

int N_layer;     // include input and output layer
int *N_neuron;
int N, M, N_max;        // input and output layer
float **x_cpu;
float **w_cpu;
float **data_test, **data_train;
float **label_test, **label_train;

float **gpu_data_test, **gpu_data_train, **gpu_label_test, **gpu_label_train;
float **x_gpu, **w_gpu, **delta_gpu;

float *curnd;
hiprandGenerator_t curand_gen;


void learning(){

	// transfer data
	random_init();
	cpu_to_gpu_matrix();
	cpu_to_gpu_data_test();

	// add noise

	for(int loop=0; loop<5000; loop++){
		cerr << "loop: " << loop << endl;
		for(int ib=0; ib<NTrain/batch_size; ib++){
			printf("loop, batch: %d, %d\n", loop, ib);
			cpu_to_gpu_data_train(ib*batch_size);
//			deform_image();

			// for each data
			for(int i=0; i<batch_size; i++){
			//	train_step
				train_step(i, loop);
			}

			// calc train error
			train_error();
		}

		// calc test error
		test_error();

		output_weight();
	}
}
int main(){
	cublasInit();

	load_data();
	gpu_alloc();
	learning();

	gpu_free();
	return 0;
}